#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#define n 1000
using namespace std;

__global__ void vectorAdd(int *a, int *b, int *result) 
{
    int tid = threadIdx.x;
    if(tid <= n) 
    {
        result[tid] = a[tid] + b[tid];
    }
}

void print_array(int *a) 
{
    for(int i=0; i<n; i++) {
        cout<<"  "<<a[i];
    }
    cout<<endl;
}

void init_array(int *a) 
{
    for(int i=0;i<n;i++) 
    {
        a[i] = rand()%100 + 1;
    }
}

int main() 
{
    int *d, *e, *res;
    int *d_par, *e_par, *res_par;
 
    d = new int[n];
    e = new int[n];
    res = new int[n];
    int size = n * sizeof(int);
    
    hipMalloc(&d_par, size);
    hipMalloc(&e_par, size);
    hipMalloc(&res_par, size);
    
    init_array(d);
    init_array(e);
    
    print_array(d);
    print_array(e);
            
    hipMemcpy(d_par, d, size, hipMemcpyHostToDevice);
    hipMemcpy(e_par, e, size, hipMemcpyHostToDevice);
    
    vectorAdd<<<1,n>>>(d_par, e_par, res_par);
    
    hipMemcpy(res, res_par, size, hipMemcpyDeviceToHost);
    
    cout<<"Results : "<<endl;
    print_array(res);
         
    hipFree(d_par);
    hipFree(e_par);
    hipFree(res_par);
        
    return 0;
}
